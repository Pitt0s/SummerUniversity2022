#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include "util.hpp"

// implement a kernel that reverses a string of length n in place
__global__
void reverse_string(char* str, int n)
{
	__shared__ char buffer[1024];
	
	int block_start = blockDim.x * blockIdx.x;
	int lid = threadIdx.x;
	int gid = lid + block_start;

	if (gid < n)
	{
		buffer[lid] = str[n-gid-1];
		str[gid] = buffer[lid];
	}
}

int main(int argc, char** argv) {
    // check that the user has passed a string to reverse
    if(argc<2) {
        std::cout << "useage : ./string_reverse \"string to reverse\"\n" << std::endl;
        exit(0);
    }

    // determine the length of the string, and copy in to buffer
    auto n = strlen(argv[1]);
    auto string = malloc_managed<char>(n+1);
    std::copy(argv[1], argv[1]+n, string);
    string[n] = 0; // add null terminator

    std::cout << "string to reverse:\n" << string << "\n";

    // call the string reverse function
	int block_dim = 128;
	int numBlock = (n-1)/block_dim + 1;
	reverse_string<<<numBlock, block_dim>>>(string, n);

    // print reversed string
    hipDeviceSynchronize();
    std::cout << "reversed string:\n" << string << "\n";

    // free memory
    hipFree(string);

    return 0;
}

